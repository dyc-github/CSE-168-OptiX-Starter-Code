#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include "Geometries.h"

using namespace optix;

rtBuffer<Triangle> triangles; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and triangle
    Triangle tri = triangles[primIndex];
    float t;

    // TODO: implement triangle intersection test here
    float3 normal = normalize(cross(tri.v2 - tri.v0, tri.v1 - tri.v0));
    
    t = (dot(tri.v0, normal) - dot(ray.origin, normal)) / dot(ray.direction, normal);
    
    //barycentric coordinates: https://cdn-uploads.piazza.com/paste/kfpn5k0uz5667e/24c3d2d5ce14011276b44c34986dfdae4bae9865111501bb37f4a24ab361e365/cse167_week4_discussion.pdf
    float3 p = t * ray.direction + ray.origin;
    //vector from v0 to the point of intersection(p)
    optix::float3 AP = p - tri.v0;
    optix::float3 AB = tri.v1 - tri.v0;
    optix::float3 AC = tri.v2 - tri.v0;
    ////dot product
    float dot00 = optix::dot(AC, AC);
    float dot01 = optix::dot(AC, AB);
    float dot0P = optix::dot(AC, AP);
    float dot11 = optix::dot(AB, AB);
    float dot1P = optix::dot(AB, AP);
    ////Compute coordinates
    float denom = dot00 * dot11 - dot01 * dot01;
    float alpha = (dot11 * dot0P - dot01 * dot1P) / denom;
    float beta = (dot00 * dot1P - dot01 * dot0P) / denom;
    float gamma = 1.0 - alpha - beta;
 

    if (alpha < 0 || alpha >= 1 || beta < 0 || beta >= 1 || gamma < 0 || gamma >= 1) {
        return;
    }

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes

        // TODO: assign attribute variables here
        attrib = tri.attributes;
        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Triangle tri = triangles[primIndex];

    // TODO: implement triangle bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}