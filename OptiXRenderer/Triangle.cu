#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include "Geometries.h"

using namespace optix;

rtBuffer<Triangle> triangles; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );
rtDeclareVariable(float3, intersectionPos, attribute intersectionPosition, );
rtDeclareVariable(float3, normal, attribute normal, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and triangle
    Triangle tri = triangles[primIndex];
    float t;

    // TODO: implement triangle intersection test here
    float3 norm = normalize(cross(tri.v2 - tri.v0, tri.v1 - tri.v0));
    
    t = (dot(tri.v0, norm) - dot(ray.origin, norm)) / dot(ray.direction, norm);
    
    //Barycentric Approach.Reference.https://ceng2.ktu.edu.tr/~cakir/files/grafikler/Texture_Mapping.pdf
    float3 p = t * ray.direction + ray.origin;
    //vector from v0 to the point of intersection(p)
    optix::float3 AP = p - tri.v0;
    optix::float3 AB = tri.v1 - tri.v0;
    optix::float3 AC = tri.v2 - tri.v0;
    ////dot product
    float dot00 = optix::dot(AC, AC);
    float dot01 = optix::dot(AC, AB);
    float dot0P = optix::dot(AC, AP);
    float dot11 = optix::dot(AB, AB);
    float dot1P = optix::dot(AB, AP);
    ////Compute coordinates
    float denom = dot00 * dot11 - dot01 * dot01;
    float alpha = (dot11 * dot0P - dot01 * dot1P) / denom;
    float beta = (dot00 * dot1P - dot01 * dot0P) / denom;
    float gamma = 1.0 - alpha - beta;
 

    if (alpha < 0 || alpha >= 1 || beta < 0 || beta >= 1 || gamma < 0 || gamma >= 1) {
        return;
    }

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes
        attrib = tri.attributes;
        intersectionPos = p;
        normal = norm;
        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Triangle tri = triangles[primIndex];

    // TODO: implement triangle bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}