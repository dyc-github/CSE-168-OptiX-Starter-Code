#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include "Geometries.h"

using namespace optix;

rtBuffer<Triangle> triangles; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and triangle
    Triangle tri = triangles[primIndex];
    float t;

    // TODO: implement triangle intersection test here
    float3 normal = normalize(cross(tri.v2 - tri.v0, tri.v1 - tri.v0));
    
    t = (dot(tri.v0, normal) - dot(ray.origin, normal)) / dot(ray.direction, normal);
    
    //barycentric coordinates: https://cdn-uploads.piazza.com/paste/kfpn5k0uz5667e/24c3d2d5ce14011276b44c34986dfdae4bae9865111501bb37f4a24ab361e365/cse167_week4_discussion.pdf
    float3 p = t * ray.direction + ray.origin;

    float alpha = (-(p.x - tri.v1.x) * (tri.v2.y - tri.v1.y) + (p.y - tri.v1.y) * (tri.v2.x - tri.v1.x))/
        (-(tri.v0.x - tri.v1.x)*(tri.v2.y - tri.v1.y) + (tri.v0.y-tri.v1.y)*(tri.v2.x - tri.v1.x));
    float beta = (-(p.x - tri.v2.x) * (tri.v0.y - tri.v2.y) + (p.y - tri.v2.y) * (tri.v0.x - tri.v2.x)) /
        (-(tri.v1.x - tri.v2.x) * (tri.v0.y - tri.v2.y) + (tri.v1.y - tri.v2.y) * (tri.v0.x - tri.v2.x));
    float gamma = 1 - alpha - beta;
    if (alpha < 0 || alpha > 1 || beta < 0 || beta > 1 || gamma < 0 || gamma > 1) {
        return;
    }

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes

        // TODO: assign attribute variables here

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Triangle tri = triangles[primIndex];

    // TODO: implement triangle bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}