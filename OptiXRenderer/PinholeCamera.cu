#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#include "Payloads.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)

rtDeclareVariable(int1, frameID, , );

// Camera info 

// TODO:: delcare camera varaibles here
rtDeclareVariable(float, height, , );
rtDeclareVariable(float, width, , );

rtDeclareVariable(float3, eye, ,);
rtDeclareVariable(float3, center, , );
rtDeclareVariable(float3, up, , );
rtDeclareVariable(float, fovy, , );





RT_PROGRAM void generateRays()
{
    float3 result = make_float3(0.f);

    // TODO: calculate the ray direction (change the following lines)
    optix::float3 w = optix::normalize(eye-center);
    optix::float3 u = optix::normalize(optix::cross(up, w));
    optix::float3 v = -optix::cross(w, u); //Im not sure why but it seems that I need to flip the up axis

    float fovx = 2 * atan(width * tan(fovy/ 2) / height);

    float alpha = tan(fovx / 2) * ((launchIndex.x + .5) - (width / 2)) / (width / 2);
    float beta = tan(fovy / 2) * ((height / 2) - (launchIndex.y + .5)) / (height / 2);

    

    float3 origin = eye; 
    float3 dir = normalize(alpha * u + beta * v - w);
    float epsilon = 0.001f; 

    // TODO: modify the following lines if you need
    // Shoot a ray to compute the color of the current pixel
    Ray ray = make_Ray(origin, dir, 0, epsilon, RT_DEFAULT_MAX);
    Payload payload;
    rtTrace(root, ray, payload);

    // Write the result
    resultBuffer[launchIndex] = payload.radiance;
}