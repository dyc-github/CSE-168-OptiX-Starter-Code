#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );
rtDeclareVariable(float3, origin, , );
rtDeclareVariable(float3, attenuation, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );
rtDeclareVariable(float3, intersectionPos, attribute intersectionPosition, );
rtDeclareVariable(float3, normal, attribute normal, );

RT_PROGRAM void closestHit()
{
    // TDOO: calculate the color using the Blinn-Phong reflection model

    float3 result = attrib.ambient + attrib.emission;
    for (int i = 0; i < plights.size(); i++) {
        PointLight plight = plights[i];
        float3 lightDir = plight.pos - intersectionPos;
        float3 half = normalize(lightDir + origin - intersectionPos);

        float3 diffuse = attrib.diffuse * max(dot(normal, lightDir), 0.0f);
        float3 specular = attrib.specular * pow(max(dot(normal, half), 0.0f), attrib.shininess);
        float3 lightIntensity = plight.color / (attenuation.x + attenuation.y * length(intersectionPos - plight.pos) + attenuation.z * pow(length(intersectionPos - plight.pos), 2));
        result += lightIntensity * (diffuse + specular);
    }
    for (int i = 0; i < dlights.size(); i++) {
        DirectionalLight dlight = dlights[i];
        float3 lightDir = dlight.dir;
        float3 half = normalize(lightDir + origin - intersectionPos);

        float3 diffuse = attrib.diffuse * max(dot(normal, lightDir), 0.0f);
        float3 specular = attrib.specular * pow(max(dot(normal, half), 0.0f), attrib.shininess);
        result += dlight.color * (diffuse + specular);
    }
    payload.radiance = result;
}