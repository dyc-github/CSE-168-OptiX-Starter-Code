#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#include "Payloads.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)

rtDeclareVariable(int1, frameID, , );

// Camera info 

// TODO:: delcare camera varaibles here
rtDeclareVariable(float1, height, , );
rtDeclareVariable(float1, width, , );

rtDeclareVariable(float3, eye, ,);
rtDeclareVariable(float3, center, , );
rtDeclareVariable(float3, up, , );
rtDeclareVariable(float1, fovy, , );





RT_PROGRAM void generateRays()
{
    float3 result = make_float3(0.f);

    // TODO: calculate the ray direction (change the following lines)
    optix::float3 w = optix::normalize(eye-center);
    optix::float3 u = optix::normalize(optix::cross(up, w));
    optix::float3 v = optix::cross(w, u);

    float fovx = 2 * atanf(width.x * tanf(fovy.x / 2) / height.x);

    float alpha = -tan(fovx / 2) * (launchIndex.y - (width.x / 2)) / (width.x / 2);
    float beta = tan(fovy.x / 2) * ((height.x / 2) - launchIndex.x) / (height.x / 2);

    

    float3 origin = eye; 
    float3 dir = normalize(alpha * u + beta * v - w);
    float epsilon = 0.001f; 

    rtPrintf("launch indices: %i %i\ndir: % f % f % f\n", launchIndex.x, launchIndex.y, dir.x, dir.y, dir.z);

    // TODO: modify the following lines if you need
    // Shoot a ray to compute the color of the current pixel
    Ray ray = make_Ray(origin, dir, 0, epsilon, RT_DEFAULT_MAX);
    Payload payload;
    rtTrace(root, ray, payload);

    // Write the result
    resultBuffer[launchIndex] = payload.radiance;
}